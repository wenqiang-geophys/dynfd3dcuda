#include <stdio.h>
#include <stdlib.h>
#include "common.h"
#include "params.h"

void *cuda_malloc(size_t len){
  void *p;
  const hipError_t err = hipMalloc(&p, len);
  if (hipSuccess == err) return p;
  //fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);
  fprintf(stderr, "Error @ %s, ", __FILE__);
  fprintf(stderr, "code: %d, reson: %s\n", err, hipGetErrorString(err));
  return 0;
}

void alloc_wave(Wave *W){
  int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;
  int nx = ni + 6;
  int ny = nj + 6;
  int nz = nk + 6;

  W-> W = (real_t *) cuda_malloc(sizeof(real_t)*nx*ny*nz*WSIZE);
  W->hW = (real_t *) cuda_malloc(sizeof(real_t)*nx*ny*nz*WSIZE);
  W->mW = (real_t *) cuda_malloc(sizeof(real_t)*nx*ny*nz*WSIZE);
  W->tW = (real_t *) cuda_malloc(sizeof(real_t)*nx*ny*nz*WSIZE);
  W->matVx2Vz = (real_t *) cuda_malloc(sizeof(real_t)*nx*ny*3*3);
  W->matVy2Vz = (real_t *) cuda_malloc(sizeof(real_t)*nx*ny*3*3);

  hipMemset(W-> W, 0, sizeof(real_t)*nx*ny*nz*WSIZE);
  hipMemset(W->hW, 0, sizeof(real_t)*nx*ny*nz*WSIZE);
  hipMemset(W->mW, 0, sizeof(real_t)*nx*ny*nz*WSIZE);
  hipMemset(W->tW, 0, sizeof(real_t)*nx*ny*nz*WSIZE);

  return;
}

void dealloc_wave(Wave W){
  CUDACHECK(hipFree(W.W));
  CUDACHECK(hipFree(W.hW));
  CUDACHECK(hipFree(W.mW));
  CUDACHECK(hipFree(W.tW));
  CUDACHECK(hipFree(W.matVx2Vz));
  CUDACHECK(hipFree(W.matVy2Vz));
  return;
}

void alloc_wave_halo(){
  int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;
  int nx = ni + 6;
  int ny = nj + 6;
  int nz = nk + 6;

  CUDACHECK(hipMalloc((real_t **) &wave_yz_send0, sizeof(real_t)*ny*nz*3*WSIZE));
  CUDACHECK(hipMalloc((real_t **) &wave_yz_send1, sizeof(real_t)*ny*nz*3*WSIZE));
  CUDACHECK(hipMalloc((real_t **) &wave_yz_recv0, sizeof(real_t)*ny*nz*3*WSIZE));
  CUDACHECK(hipMalloc((real_t **) &wave_yz_recv1, sizeof(real_t)*ny*nz*3*WSIZE));

  CUDACHECK(hipMalloc((real_t **) &wave_xz_send0, sizeof(real_t)*nx*nz*3*WSIZE));
  CUDACHECK(hipMalloc((real_t **) &wave_xz_send1, sizeof(real_t)*nx*nz*3*WSIZE));
  CUDACHECK(hipMalloc((real_t **) &wave_xz_recv0, sizeof(real_t)*nx*nz*3*WSIZE));
  CUDACHECK(hipMalloc((real_t **) &wave_xz_recv1, sizeof(real_t)*nx*nz*3*WSIZE));

  CUDACHECK(hipMalloc((real_t **) &wave_xy_send0, sizeof(real_t)*nx*ny*3*WSIZE));
  CUDACHECK(hipMalloc((real_t **) &wave_xy_send1, sizeof(real_t)*nx*ny*3*WSIZE));
  CUDACHECK(hipMalloc((real_t **) &wave_xy_recv0, sizeof(real_t)*nx*ny*3*WSIZE));
  CUDACHECK(hipMalloc((real_t **) &wave_xy_recv1, sizeof(real_t)*nx*ny*3*WSIZE));

  //int size = 3 + 2 * FSIZE;
  int size = 7 + 2 * FSIZE;
  CUDACHECK(hipMalloc((real_t **) &fault_z_send0, sizeof(real_t)*ny*3*size));
  CUDACHECK(hipMalloc((real_t **) &fault_z_send1, sizeof(real_t)*ny*3*size));
  CUDACHECK(hipMalloc((real_t **) &fault_z_recv0, sizeof(real_t)*ny*3*size));
  CUDACHECK(hipMalloc((real_t **) &fault_z_recv1, sizeof(real_t)*ny*3*size));

  CUDACHECK(hipMalloc((real_t **) &fault_y_send0, sizeof(real_t)*nz*3*size));
  CUDACHECK(hipMalloc((real_t **) &fault_y_send1, sizeof(real_t)*nz*3*size));
  CUDACHECK(hipMalloc((real_t **) &fault_y_recv0, sizeof(real_t)*nz*3*size));
  CUDACHECK(hipMalloc((real_t **) &fault_y_recv1, sizeof(real_t)*nz*3*size));

  return;
}

void alloc_fault(Fault *F){
  //int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;
  //int nx = ni + 6;
  int ny = nj + 6;
  int nz = nk + 6;
  int TP_n = hostParams.TP_n;

  F-> W = (real_t *) cuda_malloc(sizeof(real_t)*2*ny*nz*FSIZE);
  F->Ws = (real_t *) cuda_malloc(sizeof(real_t)*2*ny*nz*FSIZE);
  F->mW = (real_t *) cuda_malloc(sizeof(real_t)*2*ny*nz*FSIZE);
  F->hW = (real_t *) cuda_malloc(sizeof(real_t)*2*ny*nz*FSIZE);
  F->tW = (real_t *) cuda_malloc(sizeof(real_t)*2*ny*nz*FSIZE);
  F->T11 = (real_t *) cuda_malloc(sizeof(real_t)*7*ny*nz);
  F->T12 = (real_t *) cuda_malloc(sizeof(real_t)*7*ny*nz);
  F->T13 = (real_t *) cuda_malloc(sizeof(real_t)*7*ny*nz);
  F->mT11 = (real_t *) cuda_malloc(sizeof(real_t)*7*ny*nz);
  F->mT12 = (real_t *) cuda_malloc(sizeof(real_t)*7*ny*nz);
  F->mT13 = (real_t *) cuda_malloc(sizeof(real_t)*7*ny*nz);
  F->hT11 = (real_t *) cuda_malloc(sizeof(real_t)*7*ny*nz);
  F->hT12 = (real_t *) cuda_malloc(sizeof(real_t)*7*ny*nz);
  F->hT13 = (real_t *) cuda_malloc(sizeof(real_t)*7*ny*nz);

  //if (hostParams.Friction_type == 3){
    F->TP_T  = (real_t *) cuda_malloc(sizeof(real_t)*TP_n*nj*nk);
    F->TP_P  = (real_t *) cuda_malloc(sizeof(real_t)*TP_n*nj*nk);
    F->TP_dT = (real_t *) cuda_malloc(sizeof(real_t)*TP_n*nj*nk);
    F->TP_dP = (real_t *) cuda_malloc(sizeof(real_t)*TP_n*nj*nk);
    F->TP_hy = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
    hipMemset(F->TP_T, 0, sizeof(real_t)*TP_n*nj*nk);
    hipMemset(F->TP_P, 0, sizeof(real_t)*TP_n*nj*nk);
  //}
  F->friction = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);

  // for output
  F->str_init_x = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->str_init_y = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->str_init_z = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->T0x        = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->T0y        = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->T0z        = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->dT0x       = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->dT0y       = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->dT0z       = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->Tn         = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->Ts1        = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->Ts2        = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->tTn        = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->tTs1       = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->tTs2       = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->a          = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->b          = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->L          = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->Vw         = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->str_peak   = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->mu_d       = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->Dc         = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->C0         = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->slip       = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->hslip      = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->mslip      = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->tslip      = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->Vs1        = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->Vs2        = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->State      = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->mState     = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->hState     = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->tState     = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);

  F->rup_sensor_Dy    = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->rup_sensor_Dz    = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);
  F->rup_sensor      = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);

  F->united   = (int *)   cuda_malloc(sizeof(int)*nj*nk);
  F->faultgrid   = (int *)   cuda_malloc(sizeof(int)*nj*nk);
  F->rup_index_y  = (int *)   cuda_malloc(sizeof(int)*nj*nk);
  F->rup_index_z  = (int *)   cuda_malloc(sizeof(int)*nj*nk);
  F->flag_rup     = (int *)   cuda_malloc(sizeof(int)*nj*nk);
  F->first_rup    = (int *)   cuda_malloc(sizeof(int)*nj*nk);
  F->init_t0_flag = (int *)   cuda_malloc(sizeof(int)*nj*nk);
  F->init_t0      = (real_t *) cuda_malloc(sizeof(real_t)*nj*nk);

  hipMemset(F->init_t0_flag, 0, sizeof(int)  *nj*nk);
  hipMemset(F->init_t0,      0, sizeof(real_t)*nj*nk);
  hipMemset(F->slip,         0, sizeof(real_t)*nj*nk);
  hipMemset(F->hslip,        0, sizeof(real_t)*nj*nk);

  return;
}

void dealloc_fault(Fault F){
  CUDACHECK(hipFree(F.W           ));
  CUDACHECK(hipFree(F.Ws          ));
  CUDACHECK(hipFree(F.mW          ));
  CUDACHECK(hipFree(F.hW          ));
  CUDACHECK(hipFree(F.tW          ));
  CUDACHECK(hipFree(F.T11         ));
  CUDACHECK(hipFree(F.T12         ));
  CUDACHECK(hipFree(F.T13         ));
  CUDACHECK(hipFree(F.str_init_x  ));
  CUDACHECK(hipFree(F.str_init_y  ));
  CUDACHECK(hipFree(F.str_init_z  ));
  CUDACHECK(hipFree(F.Tn          ));
  CUDACHECK(hipFree(F.Ts1         ));
  CUDACHECK(hipFree(F.Ts2         ));
  CUDACHECK(hipFree(F.tTn         ));
  CUDACHECK(hipFree(F.tTs1        ));
  CUDACHECK(hipFree(F.tTs2        ));
  CUDACHECK(hipFree(F.a           ));
  CUDACHECK(hipFree(F.b           ));
  CUDACHECK(hipFree(F.str_peak    ));
  CUDACHECK(hipFree(F.C0          ));
  CUDACHECK(hipFree(F.slip        ));
  CUDACHECK(hipFree(F.hslip       ));
  CUDACHECK(hipFree(F.Vs1         ));
  CUDACHECK(hipFree(F.Vs2         ));
  CUDACHECK(hipFree(F.State       ));
  CUDACHECK(hipFree(F.mState      ));
  CUDACHECK(hipFree(F.hState      ));
  CUDACHECK(hipFree(F.tState      ));
  CUDACHECK(hipFree(F.rup_index_y ));
  CUDACHECK(hipFree(F.rup_index_z ));
  CUDACHECK(hipFree(F.flag_rup    ));
  CUDACHECK(hipFree(F.first_rup   ));
  CUDACHECK(hipFree(F.init_t0_flag));
  CUDACHECK(hipFree(F.init_t0     ));
  return;
}

void alloc_fault_coef(Fault *FC){
  //int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;
  //int nx = ni + 6;
  int ny = nj + 6;
  int nz = nk + 6;

  FC->rho_f = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*2);
  FC->mu_f  = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*2);
  FC->lam_f = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*2);

  FC->D11_1 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D12_1 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D13_1 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D21_1 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D22_1 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D23_1 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D31_1 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D32_1 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D33_1 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);

  FC->D11_2 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D12_2 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D13_2 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D21_2 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D22_2 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D23_2 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D31_2 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D32_2 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->D33_2 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);

  FC->matMin2Plus1 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matMin2Plus2 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matMin2Plus3 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matMin2Plus4 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matMin2Plus5 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);

  FC->matPlus2Min1 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matPlus2Min2 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matPlus2Min3 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matPlus2Min4 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matPlus2Min5 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);

  FC->matT1toVxm = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matVytoVxm = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matVztoVxm = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matT1toVxp = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matVytoVxp = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);
  FC->matVztoVxp = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3*3);

  FC->vec_n  = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3);
  FC->vec_s1 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3);
  FC->vec_s2 = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz*3);
  FC->x_et   = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz);
  FC->y_et   = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz);
  FC->z_et   = (real_t *) cuda_malloc(sizeof(real_t)*ny*nz);

  FC->matVx2Vz1     = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matVy2Vz1     = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matVx2Vz2     = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matVy2Vz2     = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matVx1_free   = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matVy1_free   = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matVx2_free   = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matVy2_free   = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matPlus2Min1f = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matPlus2Min2f = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matPlus2Min3f = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matMin2Plus1f = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matMin2Plus2f = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matMin2Plus3f = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);

  FC->matT1toVxfm = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matVytoVxfm = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matT1toVxfp = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);
  FC->matVytoVxfp = (real_t *) cuda_malloc(sizeof(real_t)*ny*3*3);

  return;
}

void dealloc_fault_coef(Fault FC){
  CUDACHECK(hipFree( FC.rho_f         ));
  CUDACHECK(hipFree( FC.mu_f          ));
  CUDACHECK(hipFree( FC.lam_f         ));
  CUDACHECK(hipFree( FC.D21_1         ));
  CUDACHECK(hipFree( FC.D22_1         ));
  CUDACHECK(hipFree( FC.D23_1         ));
  CUDACHECK(hipFree( FC.D31_1         ));
  CUDACHECK(hipFree( FC.D32_1         ));
  CUDACHECK(hipFree( FC.D33_1         ));
  CUDACHECK(hipFree( FC.D21_2         ));
  CUDACHECK(hipFree( FC.D22_2         ));
  CUDACHECK(hipFree( FC.D23_2         ));
  CUDACHECK(hipFree( FC.D31_2         ));
  CUDACHECK(hipFree( FC.D32_2         ));
  CUDACHECK(hipFree( FC.D33_2         ));
  CUDACHECK(hipFree( FC.matMin2Plus1  ));
  CUDACHECK(hipFree( FC.matMin2Plus2  ));
  CUDACHECK(hipFree( FC.matMin2Plus3  ));
  CUDACHECK(hipFree( FC.matMin2Plus4  ));
  CUDACHECK(hipFree( FC.matMin2Plus5  ));
  CUDACHECK(hipFree( FC.matPlus2Min1  ));
  CUDACHECK(hipFree( FC.matPlus2Min2  ));
  CUDACHECK(hipFree( FC.matPlus2Min3  ));
  CUDACHECK(hipFree( FC.matPlus2Min4  ));
  CUDACHECK(hipFree( FC.matPlus2Min5  ));
  CUDACHECK(hipFree( FC.vec_n         ));
  CUDACHECK(hipFree( FC.vec_s1        ));
  CUDACHECK(hipFree( FC.vec_s2        ));
  CUDACHECK(hipFree( FC.x_et          ));
  CUDACHECK(hipFree( FC.y_et          ));
  CUDACHECK(hipFree( FC.z_et          ));
  CUDACHECK(hipFree( FC.matVx2Vz1     ));
  CUDACHECK(hipFree( FC.matVy2Vz1     ));
  CUDACHECK(hipFree( FC.matVx2Vz2     ));
  CUDACHECK(hipFree( FC.matVy2Vz2     ));
  CUDACHECK(hipFree( FC.matVx1_free   ));
  CUDACHECK(hipFree( FC.matVy1_free   ));
  CUDACHECK(hipFree( FC.matVx2_free   ));
  CUDACHECK(hipFree( FC.matVy2_free   ));
  CUDACHECK(hipFree( FC.matPlus2Min1f ));
  CUDACHECK(hipFree( FC.matPlus2Min2f ));
  CUDACHECK(hipFree( FC.matPlus2Min3f ));
  CUDACHECK(hipFree( FC.matMin2Plus1f ));
  CUDACHECK(hipFree( FC.matMin2Plus2f ));
  CUDACHECK(hipFree( FC.matMin2Plus3f ));
  return;
}

void alloc_pml_host (PML *P){
  int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;
  P->Ax = (real_t*)malloc(sizeof(real_t)*ni);
  P->Bx = (real_t*)malloc(sizeof(real_t)*ni);
  P->Dx = (real_t*)malloc(sizeof(real_t)*ni);
  P->Ay = (real_t*)malloc(sizeof(real_t)*nj);
  P->By = (real_t*)malloc(sizeof(real_t)*nj);
  P->Dy = (real_t*)malloc(sizeof(real_t)*nj);
  P->Az = (real_t*)malloc(sizeof(real_t)*nk);
  P->Bz = (real_t*)malloc(sizeof(real_t)*nk);
  P->Dz = (real_t*)malloc(sizeof(real_t)*nk);
  return;
}

void dealloc_pml_host (PML P){
  free(P.Ax);
  free(P.Bx);
  free(P.Dx);
  free(P.Ay);
  free(P.By);
  free(P.Dy);
  free(P.Az);
  free(P.Bz);
  free(P.Dz);
  return;
}

void alloc_pml (PML *P){
  int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;
  int N = hostParams.PML_N;
  P->Ax = (real_t *) cuda_malloc(sizeof(real_t)*ni);
  P->Bx = (real_t *) cuda_malloc(sizeof(real_t)*ni);
  P->Dx = (real_t *) cuda_malloc(sizeof(real_t)*ni);
  P->Ay = (real_t *) cuda_malloc(sizeof(real_t)*nj);
  P->By = (real_t *) cuda_malloc(sizeof(real_t)*nj);
  P->Dy = (real_t *) cuda_malloc(sizeof(real_t)*nj);
  P->Az = (real_t *) cuda_malloc(sizeof(real_t)*nk);
  P->Bz = (real_t *) cuda_malloc(sizeof(real_t)*nk);
  P->Dz = (real_t *) cuda_malloc(sizeof(real_t)*nk);

  if(P->isx1) {
    size_t ibytes = N*nj*nk*WSIZE*sizeof(real_t);
    P-> Wx1 = (real_t *) cuda_malloc(ibytes);
    P->hWx1 = (real_t *) cuda_malloc(ibytes);
    P->mWx1 = (real_t *) cuda_malloc(ibytes);
    P->tWx1 = (real_t *) cuda_malloc(ibytes);
    hipMemset(P-> Wx1, 0, ibytes);
    hipMemset(P->hWx1, 0, ibytes);
    hipMemset(P->mWx1, 0, ibytes);
    hipMemset(P->tWx1, 0, ibytes);
  }
  if(P->isx2){
    size_t ibytes = N*nj*nk*WSIZE*sizeof(real_t);
    P-> Wx2 = (real_t *) cuda_malloc(ibytes);
    P->hWx2 = (real_t *) cuda_malloc(ibytes);
    P->mWx2 = (real_t *) cuda_malloc(ibytes);
    P->tWx2 = (real_t *) cuda_malloc(ibytes);
    hipMemset(P-> Wx2, 0, ibytes);
    hipMemset(P->hWx2, 0, ibytes);
    hipMemset(P->mWx2, 0, ibytes);
    hipMemset(P->tWx2, 0, ibytes);
  }
  if(P->isy1) {
    size_t ibytes = N*nk*ni*WSIZE*sizeof(real_t);
    P-> Wy1 = (real_t *) cuda_malloc(ibytes);
    P->hWy1 = (real_t *) cuda_malloc(ibytes);
    P->mWy1 = (real_t *) cuda_malloc(ibytes);
    P->tWy1 = (real_t *) cuda_malloc(ibytes);
    hipMemset(P-> Wy1, 0, ibytes);
    hipMemset(P->hWy1, 0, ibytes);
    hipMemset(P->mWy1, 0, ibytes);
    hipMemset(P->tWy1, 0, ibytes);
  }
  if(P->isy2){
    size_t ibytes = N*nk*ni*WSIZE*sizeof(real_t);
    P-> Wy2 = (real_t *) cuda_malloc(ibytes);
    P->hWy2 = (real_t *) cuda_malloc(ibytes);
    P->mWy2 = (real_t *) cuda_malloc(ibytes);
    P->tWy2 = (real_t *) cuda_malloc(ibytes);
    hipMemset(P-> Wy2, 0, ibytes);
    hipMemset(P->hWy2, 0, ibytes);
    hipMemset(P->mWy2, 0, ibytes);
    hipMemset(P->tWy2, 0, ibytes);
  }
  if(P->isz1) {
    size_t ibytes = N*ni*nj*WSIZE*sizeof(real_t);
    P-> Wz1 = (real_t *) cuda_malloc(ibytes);
    P->hWz1 = (real_t *) cuda_malloc(ibytes);
    P->mWz1 = (real_t *) cuda_malloc(ibytes);
    P->tWz1 = (real_t *) cuda_malloc(ibytes);
    hipMemset(P-> Wz1, 0, ibytes);
    hipMemset(P->hWz1, 0, ibytes);
    hipMemset(P->mWz1, 0, ibytes);
    hipMemset(P->tWz1, 0, ibytes);
  }
  if(P->isz2){
    size_t ibytes = N*ni*nj*WSIZE*sizeof(real_t);
    P-> Wz2 = (real_t *) cuda_malloc(ibytes);
    P->hWz2 = (real_t *) cuda_malloc(ibytes);
    P->mWz2 = (real_t *) cuda_malloc(ibytes);
    P->tWz2 = (real_t *) cuda_malloc(ibytes);
    hipMemset(P-> Wz2, 0, ibytes);
    hipMemset(P->hWz2, 0, ibytes);
    hipMemset(P->mWz2, 0, ibytes);
    hipMemset(P->tWz2, 0, ibytes);
  }
  return;
}

void dealloc_pml (PML P){
  CUDACHECK(hipFree( P.Ax ));
  CUDACHECK(hipFree( P.Ay ));
  CUDACHECK(hipFree( P.Az ));
  CUDACHECK(hipFree( P.Bx ));
  CUDACHECK(hipFree( P.By ));
  CUDACHECK(hipFree( P.Bz ));
  CUDACHECK(hipFree( P.Dx ));
  CUDACHECK(hipFree( P.Dy ));
  CUDACHECK(hipFree( P.Dz ));

  if(P.isx1) {
    CUDACHECK(hipFree( P. Wx1 ));
    CUDACHECK(hipFree( P.hWx1 ));
    CUDACHECK(hipFree( P.mWx1 ));
    CUDACHECK(hipFree( P.tWx1 ));
  }
  if(P.isx2){
    CUDACHECK(hipFree( P. Wx2 ));
    CUDACHECK(hipFree( P.hWx2 ));
    CUDACHECK(hipFree( P.mWx2 ));
    CUDACHECK(hipFree( P.tWx2 ));
  }
  if(P.isy1) {
    CUDACHECK(hipFree( P. Wy1 ));
    CUDACHECK(hipFree( P.hWy1 ));
    CUDACHECK(hipFree( P.mWy1 ));
    CUDACHECK(hipFree( P.tWy1 ));
  }
  if(P.isy2){
    CUDACHECK(hipFree( P. Wy2 ));
    CUDACHECK(hipFree( P.hWy2 ));
    CUDACHECK(hipFree( P.mWy2 ));
    CUDACHECK(hipFree( P.tWy2 ));
  }
  if(P.isz1) {
    CUDACHECK(hipFree( P. Wz1 ));
    CUDACHECK(hipFree( P.hWz1 ));
    CUDACHECK(hipFree( P.mWz1 ));
    CUDACHECK(hipFree( P.tWz1 ));
  }
  if(P.isz2){
    CUDACHECK(hipFree( P. Wz2 ));
    CUDACHECK(hipFree( P.hWz2 ));
    CUDACHECK(hipFree( P.mWz2 ));
    CUDACHECK(hipFree( P.tWz2 ));
  }
  return;
}

void cpy_host2device_pml(PML P, const PML h_P){
  int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;
  hipMemcpy(P.Ax, h_P.Ax, sizeof(real_t)*ni, hipMemcpyHostToDevice);
  hipMemcpy(P.Bx, h_P.Bx, sizeof(real_t)*ni, hipMemcpyHostToDevice);
  hipMemcpy(P.Dx, h_P.Dx, sizeof(real_t)*ni, hipMemcpyHostToDevice);
  hipMemcpy(P.Ay, h_P.Ay, sizeof(real_t)*nj, hipMemcpyHostToDevice);
  hipMemcpy(P.By, h_P.By, sizeof(real_t)*nj, hipMemcpyHostToDevice);
  hipMemcpy(P.Dy, h_P.Dy, sizeof(real_t)*nj, hipMemcpyHostToDevice);
  hipMemcpy(P.Az, h_P.Az, sizeof(real_t)*nk, hipMemcpyHostToDevice);
  hipMemcpy(P.Bz, h_P.Bz, sizeof(real_t)*nk, hipMemcpyHostToDevice);
  hipMemcpy(P.Dz, h_P.Dz, sizeof(real_t)*nk, hipMemcpyHostToDevice);
  return;
}
