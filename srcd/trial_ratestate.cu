#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "common.h"
#include "params.h"
#include "macdrp.h"


//#define DEBUG
//extern __device__ __constant__ int Flags[8][3];
extern __device__ real_t norm3(real_t *A);
extern __device__ real_t dot_product(real_t *A, real_t *B);
extern __device__ void matmul3x1(real_t A[][3], real_t B[3], real_t C[3]);
extern __device__ void matmul3x3(real_t A[][3], real_t B[][3], real_t C[][3]);
extern __device__ void invert3x3(real_t m[][3]);
extern __device__ real_t Fr_func(const real_t r, const real_t R);
extern __device__ real_t Gt_func(const real_t t, const real_t T);

#define DOT3(A,B) (A[0]*B[0]+A[1]*B[1]+A[2]*B[2])

__device__ void NRsearch(double *xnew, int *iter, double *err,
    double x, double psi, double RS_a, double Vhat, double dt, double coef, int verbose){
  int iter_max = 100;
  double eps = 1e-12;

  double RS_V0 = 1e-6;
  //double Tn = 120e6;
  double f, df, d;

  *iter = 0;
  *xnew = x; // initial guess
  while (*iter < iter_max) {

    //if(verbose){
    //  printf("x = %e, V = %e, psi = %e, RS_a = %e, Vhat = %e, dt = %e, coef = %e\n",
    //      x, 2.0*RS_V0*exp(-psi/RS_a)*sinh(x), psi, RS_a, Vhat, dt, coef);
    //}
    //f = 2.0 * RS_V0 * exp(-psi/RS_a) * sinh(x) + dt * coef * Tn * RS_a * x - Vhat;
    //df = 2.0 * RS_V0 * exp(-psi/RS_a) * cosh(x) + dt * coef * Tn * RS_a;
    f  = 2.0 * exp(-psi/RS_a) * RS_V0 * sinh(x) + coef * x - Vhat;
    df = 2.0 * exp(-psi/RS_a) * RS_V0 * cosh(x) + coef;

    d = -f/(df+1e-100);
    *xnew = x + d;

    *err = fabs(d)/(fabs(x) + 1e-100);

    if(verbose){
      printf("iter = %d, f = %e, df = %e, f/df = %e, x = %e\n",
          *iter, f, df, d, *xnew);
    }

    x = *xnew; (*iter)++;

    if (*err < eps) break;
  }

}

__global__ void trial_rs_cu(Wave w, Fault f, real_t *M,
    int it, int irk, int FlagX, int FlagY, int FlagZ)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int k = blockIdx.y * blockDim.y + threadIdx.y;

  int j1 = j + 3;
  int k1 = k + 3;

  int ni = par.NX / par.PX;
  int nj = par.NY / par.PY;
  int nk = par.NZ / par.PZ;

  int nx = ni + 6;
  int ny = nj + 6;
  int nz = nk + 6;

  int nxyz = nx * ny * nz;
  int nyz = ny * nz;
  int nyz2 = 2 * ny * nz;

  real_t *XIX = M;
  real_t *XIY = XIX + nxyz;
  real_t *XIZ = XIY + nxyz;
  real_t *JAC = M + 9 * nxyz;
  //real_t *RHO = M + 12 * nxyz;

  //stride = nx * ny * nz; // x vary first
  //real_t *w_Vx  = w.W + 0 * nxyz;
  //real_t *w_Vy  = w.W + 1 * nxyz;
  //real_t *w_Vz  = w.W + 2 * nxyz;
  real_t *w_Txx = w.W + 3 * nxyz;
  real_t *w_Tyy = w.W + 4 * nxyz;
  real_t *w_Tzz = w.W + 5 * nxyz;
  real_t *w_Txy = w.W + 6 * nxyz;
  real_t *w_Txz = w.W + 7 * nxyz;
  real_t *w_Tyz = w.W + 8 * nxyz;

  // Split nodes
  //stride = ny * nz * 2; // y vary first
  real_t *f_Vx  = f.W + 0 * nyz2;
  real_t *f_Vy  = f.W + 1 * nyz2;
  real_t *f_Vz  = f.W + 2 * nyz2;
  real_t *f_T21 = f.W + 3 * nyz2;
  real_t *f_T22 = f.W + 4 * nyz2;
  real_t *f_T23 = f.W + 5 * nyz2;
  real_t *f_T31 = f.W + 6 * nyz2;
  real_t *f_T32 = f.W + 7 * nyz2;
  real_t *f_T33 = f.W + 8 * nyz2;

  real_t *f_mVx  = f.mW + 0 * nyz2;
  real_t *f_mVy  = f.mW + 1 * nyz2;
  real_t *f_mVz  = f.mW + 2 * nyz2;

  real_t *f_tVx  = f.tW + 0 * nyz2;
  real_t *f_tVy  = f.tW + 1 * nyz2;
  real_t *f_tVz  = f.tW + 2 * nyz2;

  real_t DH = par.DH;
  real_t rDH = 1.0/DH;
  real_t DT = par.DT;

  //int istep = it % 8;
  //int sign1 = irk % 2;
  //int FlagX = Flags[istep][0];
  //int FlagY = Flags[istep][1];
  //int FlagZ = Flags[istep][2];
  int i0 = nx/2;

  real_t xix, xiy, xiz;
  //real_t etx, ety, etz;
  //real_t ztx, zty, ztz;
  real_t jac;
  real_t vec_n0;
  real_t jacvec;
  //real_t lam, mu;
  //real_t lam2mu;
  //real_t rrho;
  real_t rho;

  real_t Mrho[2], Rx[2], Ry[2], Rz[2];
  real_t R1[2], R2[2], R3[2];
  real_t DH2 = DH*DH;
  real_t T11, T12, T13;
  int i;
  int pos, pos_f, pos1;
  //int slice;
  real_t DyT21, DyT22, DyT23;
  real_t DzT31, DzT32, DzT33;
  real_t vecT31[7];
  real_t vecT32[7];
  real_t vecT33[7];

  //if(sign1) { FlagX *= -1; FlagY *= -1; FlagZ *= -1; }

  /* check RK flag */
  //if(100==j && 100 == k)printf("(%d, %d):%+d%+d%+d\n", it, irk, FlagX, FlagY, FlagZ);

//#ifdef FreeSurface
//  if ( j >= 30+3 && j < ny-31-3 && k >=30+3 && k < nz-3){  // not united
//#else
//  if ( j >= 30+3 && j < ny-31-3 && k >=30+3 && k < nz-31-3){  // not united
//#endif
//#ifdef FreeSurface
//  if ( j >= 30 && j < nj-31 && k >=30 && k < nk){  // not united
//#else
//  if ( j >= 30 && j < nj-31 && k >=30 && k < nk-31){  // not united
//#endif
  if ( j < nj && k < nk && f.united[j + k * nj] == 0){  // not united

    pos1 = j1 + k1 * ny + 3*nyz;
    if(irk == 0){
      f.mT11[pos1] = f.T11[pos1];
      f.mT12[pos1] = f.T12[pos1];
      f.mT13[pos1] = f.T13[pos1];
    }
    //km = NZ - (thisid[2]*nk+k-3);
    //int km = (nz - 6) - (k-3); // nk2-3, nk2-2, nk2-1 ==> (3, 2, 1)
    int km = nk - k; // nk2-3, nk2-2, nk2-1 ==> (3, 2, 1)

    real_t vec_n[3];
    real_t vec_s1[3];
    real_t vec_s2[3];

    pos = (j1 + k1 * ny) * 3;
    vec_s1[0] = f.vec_s1[pos + 0];
    vec_s1[1] = f.vec_s1[pos + 1];
    vec_s1[2] = f.vec_s1[pos + 2];
    vec_s2[0] = f.vec_s2[pos + 0];
    vec_s2[1] = f.vec_s2[pos + 1];
    vec_s2[2] = f.vec_s2[pos + 2];

    pos = j1 + k1 * ny + i0 * ny * nz;
    vec_n[0] = XIX[pos];
    vec_n[1] = XIY[pos];
    vec_n[2] = XIZ[pos];
    vec_n0 = norm3(vec_n);
    jacvec = JAC[pos] * vec_n0;
    for (int ii = 0; ii < 3; ii++){
      vec_n[ii] /= vec_n0;
    }

    //vec_n [0] = 1;vec_n [1] = 0;vec_n [2] = 0;
    //vec_s1[0] = 0;vec_s1[1] = 1;vec_s1[2] = 0;
    //vec_s2[0] = 0;vec_s2[1] = 0;vec_s2[2] = 1;

    for (int m = 0; m < 2; m++){

      i = i0 + 2*m - 1; // i0-1, i0+1
      //pos_m = (i*ny*nz+j1*nz+k1)*MSIZE;
      //pos   = (i*ny*nz+j1*nz+k1)*WSIZE;
      //pos = k1 * ny * nx + j1 * nx + i;
      //pos = i * ny * nz + j1 * nz + k1;
      pos = j1 + k1 * ny + i * ny * nz;

      xix = XIX[pos];
      xiy = XIY[pos];
      xiz = XIZ[pos];
      jac = JAC[pos];
      //rho = RHO[pos];
      // bimaterial
      rho = f.rho_f[j1 + k1 * ny + m * ny * nz];

      //xix = M[pos_m + 0]; xiy = M[pos_m + 1]; xiz = M[pos_m + 2];
      //jac = M[pos_m + 9];
      //rho = M[pos_m + 12];

      // {Txx 3} {Tyy 4} {Tzz 5} {Txy 6} {Txz 7} {Tyz 8}
      // T1 is continuous!
      //T11 = jac*(xix * w.W[pos + 3] + xiy * w.W[pos + 6] + xiz * w.W[pos + 7]);
      //T12 = jac*(xix * w.W[pos + 6] + xiy * w.W[pos + 4] + xiz * w.W[pos + 8]);
      //T13 = jac*(xix * w.W[pos + 7] + xiy * w.W[pos + 8] + xiz * w.W[pos + 5]);
      //!!T11 = jac*(xix * w_Txx[pos] + xiy * w_Txy[pos] + xiz * w_Txz[pos]);
      //!!T12 = jac*(xix * w_Txy[pos] + xiy * w_Tyy[pos] + xiz * w_Tyz[pos]);
      //!!T13 = jac*(xix * w_Txz[pos] + xiy * w_Tyz[pos] + xiz * w_Tzz[pos]);
      pos_f = j1 + k1 * ny;
      for (int l = 1; l <= 3; l++){
        pos = j1 + k1*ny + (i0+(2*m-1)*l)*ny*nz;
        xix = XIX[pos];
        xiy = XIY[pos];
        xiz = XIZ[pos];
        jac = JAC[pos];
        T11 = jac*(xix * w_Txx[pos] + xiy * w_Txy[pos] + xiz * w_Txz[pos]);
        T12 = jac*(xix * w_Txy[pos] + xiy * w_Tyy[pos] + xiz * w_Tyz[pos]);
        T13 = jac*(xix * w_Txz[pos] + xiy * w_Tyz[pos] + xiz * w_Tzz[pos]);
        f.T11[(3+(2*m-1)*l)*nyz + pos_f] = T11;
        f.T12[(3+(2*m-1)*l)*nyz + pos_f] = T12;
        f.T13[(3+(2*m-1)*l)*nyz + pos_f] = T13;
      }

      // 0 or 2 ==> i0-1 or i0+1
      //f.T11[(2*m)*ny*nz+j1*nz+k1] = T11;
      //f.T12[(2*m)*ny*nz+j1*nz+k1] = T12;
      //f.T13[(2*m)*ny*nz+j1*nz+k1] = T13;
      //pos_f = j1 + k1 * ny;
      //f.T11[(2*m)*nyz + pos_f] = T11;
      //f.T12[(2*m)*nyz + pos_f] = T12;
      //f.T13[(2*m)*nyz + pos_f] = T13;

      // {T21 3} {T22 4} {T23 5} {T31 6} {T32 7} {T33 8}
      //slice = nz*FSIZE;
      // bug fixed
      //pos_f = (m*ny*nz+j1*nz+k1)*FSIZE;
      //pos_f = k1 * ny + j1;
      real_t *t21 = f_T21 + m*nyz;
      real_t *t22 = f_T22 + m*nyz;
      real_t *t23 = f_T23 + m*nyz;
#ifdef RupSensor
      if(f.rup_sensor[j + k * nj] > par.RupThres){
#else
      if(f.rup_index_y[j + k * nj] % 7){
#endif
        //DyT21 = L22(f.W, (pos_f + 3), slice, FlagY) / DH;
        //DyT22 = L22(f.W, (pos_f + 4), slice, FlagY) / DH;
        //DyT23 = L22(f.W, (pos_f + 5), slice, FlagY) / DH;
        DyT21 = L22(t21, pos_f, 1, FlagY) * rDH;
        DyT22 = L22(t22, pos_f, 1, FlagY) * rDH;
        DyT23 = L22(t23, pos_f, 1, FlagY) * rDH;
      }else{
        DyT21 = L(t21, pos_f, 1, FlagY) * rDH;
        DyT22 = L(t22, pos_f, 1, FlagY) * rDH;
        DyT23 = L(t23, pos_f, 1, FlagY) * rDH;
        //DyT21 = L(f.W, (pos_f + 3), slice, FlagY) / DH;
        //DyT22 = L(f.W, (pos_f + 4), slice, FlagY) / DH;
        //DyT23 = L(f.W, (pos_f + 5), slice, FlagY) / DH;
      }

      for (int l = -3; l <=3 ; l++){
        //vecT31[l+3] = F->T31[m][j][k+l];
        //vecT32[l+3] = F->T32[m][j][k+l];
        //vecT33[l+3] = F->T33[m][j][k+l];
        //pos = (m*ny*nz + j1*nz + k1+l)*FSIZE;
        pos = j1 + (k1 + l) * ny;
        vecT31[l+3] = f_T31[pos + m*nyz];//f.W[pos + 6];
        vecT32[l+3] = f_T32[pos + m*nyz];//f.W[pos + 7];
        vecT33[l+3] = f_T33[pos + m*nyz];//f.W[pos + 8];
      }

      if(par.freenode && km<=3){
        ///extendvect(vecT31,km+2,0.0);
        ///extendvect(vecT32,km+2,0.0);
        ///extendvect(vecT33,km+2,0.0);
        vecT31[km+2] = 0.0;
        vecT32[km+2] = 0.0;
        vecT33[km+2] = 0.0;
        for (int l = km+3; l<7; l++){
          vecT31[l] = -vecT31[2*(km+2)-l];
          vecT32[l] = -vecT32[2*(km+2)-l];
          vecT33[l] = -vecT33[2*(km+2)-l];
        }
      } // end par.freenode
      //else
      //{
      //  DzT31 = L(f.W, (pos_f + 6), segment, FlagZ) / DH;
      //  DzT32 = L(f.W, (pos_f + 7), segment, FlagZ) / DH;
      //  DzT33 = L(f.W, (pos_f + 8), segment, FlagZ) / DH;
      //}
#ifdef RupSensor
      if(f.rup_sensor[j + k * nj] > par.RupThres){
#else
      if(f.rup_index_z[j + k * nj] % 7){
#endif
        DzT31 = vec_L22(vecT31, 3, FlagZ) * rDH;
        DzT32 = vec_L22(vecT32, 3, FlagZ) * rDH;
        DzT33 = vec_L22(vecT33, 3, FlagZ) * rDH;
      }else{
        DzT31 = vec_L(vecT31, 3, FlagZ) * rDH;
        DzT32 = vec_L(vecT32, 3, FlagZ) * rDH;
        DzT33 = vec_L(vecT33, 3, FlagZ) * rDH;
      }

      T11 = f.T11[(3+2*m-1)*nyz+pos_f];
      T12 = f.T12[(3+2*m-1)*nyz+pos_f];
      T13 = f.T13[(3+2*m-1)*nyz+pos_f];

      Rx[m] = 0.5f*( (2*m-1)*T11 + (DyT21 + DzT31)*DH )*DH2;
      Ry[m] = 0.5f*( (2*m-1)*T12 + (DyT22 + DzT32)*DH )*DH2;
      Rz[m] = 0.5f*( (2*m-1)*T13 + (DyT23 + DzT33)*DH )*DH2;

#ifdef TractionImg
      if (m == 0){ // "-" side
        Rx[m] =
          a_1 * f.T11[2*nyz+pos_f] +
          a_2 * f.T11[1*nyz+pos_f] +
          a_3 * f.T11[0*nyz+pos_f] ;
        Ry[m] =
          a_1 * f.T12[2*nyz+pos_f] +
          a_2 * f.T12[1*nyz+pos_f] +
          a_3 * f.T12[0*nyz+pos_f] ;
        Rz[m] =
          a_1 * f.T13[2*nyz+pos_f] +
          a_2 * f.T13[1*nyz+pos_f] +
          a_3 * f.T13[0*nyz+pos_f] ;
      }else{ // "+" side
        Rx[m] =
          a_1 * f.T11[4*nyz+pos_f] +
          a_2 * f.T11[5*nyz+pos_f] +
          a_3 * f.T11[6*nyz+pos_f] ;
        Ry[m] =
          a_1 * f.T12[4*nyz+pos_f] +
          a_2 * f.T12[5*nyz+pos_f] +
          a_3 * f.T12[6*nyz+pos_f] ;
        Rz[m] =
          a_1 * f.T13[4*nyz+pos_f] +
          a_2 * f.T13[5*nyz+pos_f] +
          a_3 * f.T13[6*nyz+pos_f] ;
      }

      Rx[m] = 0.5f*( (2*m-1)*Rx[m] + (DyT21 + DzT31)*DH )*DH2;
      Ry[m] = 0.5f*( (2*m-1)*Ry[m] + (DyT22 + DzT32)*DH )*DH2;
      Rz[m] = 0.5f*( (2*m-1)*Rz[m] + (DyT23 + DzT33)*DH )*DH2;
#endif

      Mrho[m] = 0.5f*jac*rho*DH*DH2;

      R1[m] = vec_n [0]*Rx[m]+vec_n [1]*Ry[m]+vec_n [2]*Rz[m];
      R2[m] = vec_s1[0]*Rx[m]+vec_s1[1]*Ry[m]+vec_s1[2]*Rz[m];
      R3[m] = vec_s2[0]*Rx[m]+vec_s2[1]*Ry[m]+vec_s2[2]*Rz[m];

    } // end m

    //========================================================================
    // add perturbation
    //========================================================================
    int gj = par.ranky * nj + j;
    int gk = par.rankz * nk + k;
    real_t y1 = (gj-par.NY/2);
#ifdef FreeSurface
    real_t z1 = (par.NZ-1-gk) - (int)(7.5e3/DH);
#else
    real_t z1 = (gk-par.NZ/2);
#endif

    real_t r = sqrt(y1*y1+z1*z1) * DH;
    real_t dTau0 = -25e6;
    real_t Tau_ini = -75e6;
#if defined TPV103 || defined TPV104
    dTau0 = -45.0e6;
    Tau_ini = -40.0e6;
#endif
    real_t t;
    if(irk == 0){
      t = it*DT;
    }else if (irk == 1 || irk == 2){
      t = (it+0.5)*DT;
    }else{
      t = (it+1)*DT;
    }

    real_t Gt = Gt_func(t, 1.0);
    Gt = Gt_func(t, par.smooth_load_T);
    real_t Fr = Fr_func(r, 3.0e3);

    pos = j + k * nj;
    f.str_init_x[pos] = -120e6;
    f.str_init_y[pos] = dTau0*Fr*Gt + Tau_ini;
    f.str_init_z[pos] = 0.0;

    if (1 == par.INPORT_STRESS_TYPE){
      // overwrite by input stress data
      f.str_init_x[pos] = f.T0x[pos] + Gt * f.dT0x[pos];
      f.str_init_y[pos] = f.T0y[pos] + Gt * f.dT0y[pos];
      f.str_init_z[pos] = f.T0z[pos] + Gt * f.dT0z[pos];
    }

    real_t T0[3];
    real_t T0_local[3];

    T0[0] = f.str_init_x[pos];
    T0[1] = f.str_init_y[pos];
    T0[2] = f.str_init_z[pos];

    T0_local[0] = vec_n [0]*T0[0]+vec_n [1]*T0[1]+vec_n [2]*T0[2];
    T0_local[1] = vec_s1[0]*T0[0]+vec_s1[1]*T0[1]+vec_s1[2]*T0[2];
    T0_local[2] = vec_s2[0]*T0[0]+vec_s2[1]*T0[1]+vec_s2[2]*T0[2];


    //T0_local[0] = -120e6;
    //T0_local[1] = dTau0*Fr_func(r, 3.0e3)*Gt_func(t, 1.0f) + Tau_ini;
    //T0_local[2] = 0.0;
    //f.str_init_y[pos] = dTau0*Fr_func(r, 3.0e3) + Tau_ini;
    //if(j==nj/2 && k==nk/2)
    //  printf("it = %03d, irk = %d, str_init_y = %e\n",
    //      it, irk, f.str_init_y[pos]);

    //========================================================================
    //========================================================================
    //real_t c1[3];
    //c1[0] = (Rx[1] - Rx[0]) / Mrho[0];
    //c1[1] = (Ry[1] - Ry[0]) / Mrho[0];
    //c1[2] = (Rz[1] - Rz[0]) / Mrho[0];
    //c1[0] = (Rx[1]/Mrho[1] - Rx[0]/Mrho[0]);
    //c1[1] = (Ry[1]/Mrho[1] - Ry[0]/Mrho[0]);
    //c1[2] = (Rz[1]/Mrho[1] - Rz[0]/Mrho[0]);

    //real_t c2 = DH * DH / Mrho[0];

    pos = j1 + k1 * ny;

    double dVx = f_mVx[pos + nyz] - f_mVx[pos];
    double dVy = f_mVy[pos + nyz] - f_mVy[pos];
    double dVz = f_mVz[pos + nyz] - f_mVz[pos];
#ifdef RKtrial
    if (irk == 3){
      dVx = f_tVx[pos + nyz] - f_tVx[pos];
      dVy = f_tVy[pos + nyz] - f_tVy[pos];
      dVz = f_tVz[pos + nyz] - f_tVz[pos];
    }
#endif
    double dt1 = DT;
#ifdef RKtrial
    if (irk == 0 || irk == 1){
      dt1 *= 0.5;
    }else if (irk == 3){
      dt1 /= 6.0;
    }
#endif

    //real_t V = f.hslip[pos] + 1e-12;
    // rate
    //real_t V = sqrt(dVx * dVx + dVy * dVy + dVz * dVz);
    real_t V;// = fabs(dVy);
    double Ratex = dVx;
    double Ratey = dVy;
    double Ratez = dVz;
    double Vini = par.Vini;
#if defined TPV103 || defined TPV104
    Ratex = dVx;
    Ratey = dVy-1e-16;
    Ratez = dVz;
#endif
#if defined TPV101 || defined TPV102
    Ratex = dVx;
    Ratey = dVy-1e-12;
    Ratez = dVz;
#endif

    double Rate_local[3];
    Rate_local[0] = vec_n [0]*Ratex+vec_n [1]*Ratey+vec_n [2]*Ratez;
    Rate_local[1] = vec_s1[0]*Ratex+vec_s1[1]*Ratey+vec_s1[2]*Ratez;
    Rate_local[2] = vec_s2[0]*Ratex+vec_s2[1]*Ratey+vec_s2[2]*Ratez;

    Rate_local[1] -= Vini;

    //real_t mRate = sqrt(Rate_local[1]*Rate_local[1]+Rate_local[2]*Rate_local[2]);

   // Rate_local[0] = 0;

#ifdef TractionImg
    double a0p,a0m;
    if(FlagX==FWD){
      a0p = a_0pF;
      a0m = a_0mF;
    }else{
      a0p = a_0pB;
      a0m = a_0mB;
    }
#endif
    //real_t dVhat_x = dVx + DT * ( c1[0] + c2 * f.str_init_x[j + k * nj]);
    //real_t dVhat_y = dVy + DT * ( c1[1] + c2 * f.str_init_y[j + k * nj]);
    //real_t dVhat_z = dVz + DT * ( c1[2] + c2 * f.str_init_z[j + k * nj]);

    double Vhat_x = dVx + dt1 * ((Rx[1]/Mrho[1] - Rx[0]/Mrho[0]) + (2.0*f.str_init_x[j + k * nj])/(jac*rho*DH));
    double Vhat_y = dVy + dt1 * ((Ry[1]/Mrho[1] - Ry[0]/Mrho[0]) + (2.0*f.str_init_y[j + k * nj])/(jac*rho*DH));
    double Vhat_z = dVz + dt1 * ((Rz[1]/Mrho[1] - Rz[0]/Mrho[0]) + (2.0*f.str_init_z[j + k * nj])/(jac*rho*DH));

    //real_t Vhat_x = fabs(dVhat_x);
    //real_t Vhat_y = fabs(dVhat_y);
    //real_t Vhat_z = fabs(dVhat_z);
    //Vhat_x = (dVhat_x);
    //Vhat_y = (dVhat_y);
    //Vhat_z = (dVhat_z);

    //real_t Vhat  = sqrt(dVhat_y * dVhat_y + dVhat_z * dVhat_z);
    //real_t vec_n[3];
    //real_t vec_s1[3];
    //real_t vec_s2[3];

    //pos = (j1 + k1 * ny) * 3;
    //vec_s1[0] = f.vec_s1[pos + 0];
    //vec_s1[1] = f.vec_s1[pos + 1];
    //vec_s1[2] = f.vec_s1[pos + 2];
    //vec_s2[0] = f.vec_s2[pos + 0];
    //vec_s2[1] = f.vec_s2[pos + 1];
    //vec_s2[2] = f.vec_s2[pos + 2];

    //pos = j1 + k1 * ny + i0 * ny * nz;
    //vec_n[0] = XIX[pos];
    //vec_n[1] = XIY[pos];
    //vec_n[2] = XIZ[pos];
    //vec_n0 = norm3(vec_n);
    //jacvec = JAC[pos] * vec_n0;
    //for (int ii = 0; ii < 3; ii++)
    //  vec_n[ii] /= vec_n0;



    double Ttilde[3];
    Ttilde[0] = f.str_init_x[j+k*nj]+2.0*(Mrho[0]*Mrho[1]*dVx/dt1+Mrho[0]*Rx[1]-Mrho[1]*Rx[0])/(DH2*(Mrho[0]+Mrho[1])*jacvec);
    Ttilde[1] = f.str_init_y[j+k*nj]+2.0*(Mrho[0]*Mrho[1]*dVy/dt1+Mrho[0]*Ry[1]-Mrho[1]*Ry[0])/(DH2*(Mrho[0]+Mrho[1])*jacvec);
    Ttilde[2] = f.str_init_z[j+k*nj]+2.0*(Mrho[0]*Mrho[1]*dVz/dt1+Mrho[0]*Rz[1]-Mrho[1]*Rz[0])/(DH2*(Mrho[0]+Mrho[1])*jacvec);

    double Ttilde_local[3];
    Ttilde_local[0] = vec_n [0]*Ttilde[0]+vec_n [1]*Ttilde[1]+vec_n [2]*Ttilde[2];
    Ttilde_local[1] = vec_s1[0]*Ttilde[0]+vec_s1[1]*Ttilde[1]+vec_s1[2]*Ttilde[2];
    Ttilde_local[2] = vec_s2[0]*Ttilde[0]+vec_s2[1]*Ttilde[1]+vec_s2[2]*Ttilde[2];

    Ttilde_local[0] = T0_local[0]+2.0*(Mrho[0]*Mrho[1]*Rate_local[0]/dt1+Mrho[0]*R1[1]-Mrho[1]*R1[0])/(DH2*(Mrho[0]+Mrho[1])*jacvec);
    Ttilde_local[1] = T0_local[1]+2.0*(Mrho[0]*Mrho[1]*Rate_local[1]/dt1+Mrho[0]*R2[1]-Mrho[1]*R2[0])/(DH2*(Mrho[0]+Mrho[1])*jacvec);
    Ttilde_local[2] = T0_local[2]+2.0*(Mrho[0]*Mrho[1]*Rate_local[2]/dt1+Mrho[0]*R3[1]-Mrho[1]*R3[0])/(DH2*(Mrho[0]+Mrho[1])*jacvec);
#ifdef TractionImg
    Ttilde_local[0] = T0_local[0]-2.0*(Mrho[0]*Mrho[1]*Rate_local[0]/dt1+Mrho[0]*R1[1]-Mrho[1]*R1[0])/(DH2*(Mrho[0]*a0p-a0m*Mrho[1])*jacvec);
    Ttilde_local[1] = T0_local[1]-2.0*(Mrho[0]*Mrho[1]*Rate_local[1]/dt1+Mrho[0]*R2[1]-Mrho[1]*R2[0])/(DH2*(Mrho[0]*a0p-a0m*Mrho[1])*jacvec);
    Ttilde_local[2] = T0_local[2]-2.0*(Mrho[0]*Mrho[1]*Rate_local[2]/dt1+Mrho[0]*R3[1]-Mrho[1]*R3[0])/(DH2*(Mrho[0]*a0p-a0m*Mrho[1])*jacvec);
#endif

    double Vtilde[3];
    Vtilde[0] = Vhat_x;
    Vtilde[1] = Vhat_y;
    Vtilde[2] = Vhat_z;

    double Vtilde_local[3];
    Vtilde_local[0] = vec_n [0]*Vtilde[0]+vec_n [1]*Vtilde[1]+vec_n [2]*Vtilde[2];
    Vtilde_local[1] = vec_s1[0]*Vtilde[0]+vec_s1[1]*Vtilde[1]+vec_s1[2]*Vtilde[2];
    Vtilde_local[2] = vec_s2[0]*Vtilde[0]+vec_s2[1]*Vtilde[1]+vec_s2[2]*Vtilde[2];

    double c = 0.5*dt1*DH2*jacvec/Mrho[0] + 0.5*dt1*DH2*jacvec/Mrho[1];
    //c = 2.0*DT*jacvec/(jac*rho*DH);
#ifdef TractionImg
    //c = -0.5*DT*DH2*jacvec*(a0p/Mrho[1]-a0m/Mrho[0]);
    c = -(a0p-a0m)*dt1*jacvec/(jac*rho*DH);
#endif

    Vtilde_local[0] = c*Ttilde_local[0];
    Vtilde_local[1] = c*Ttilde_local[1];
    Vtilde_local[2] = c*Ttilde_local[2];

    Vtilde_local[0] = c*T0_local[0]+Rate_local[0]+dt1*(R1[1]/Mrho[1]-R1[0]/Mrho[0]);
    Vtilde_local[1] = c*T0_local[1]+Rate_local[1]+dt1*(R2[1]/Mrho[1]-R2[0]/Mrho[0]);
    Vtilde_local[2] = c*T0_local[2]+Rate_local[2]+dt1*(R3[1]/Mrho[1]-R3[0]/Mrho[0]);

    //Rate_local[0] = Ratex;
    //Rate_local[1] = Ratey;
    //Rate_local[2] = Ratez;

    pos = j + k * nj;
    //double V = f.hslip[pos];
    double state = f.State[pos];

    double RS_a = f.a[pos];
    double RS_b = f.b[pos];
    double RS_f0 = 0.6;
    double RS_V0 = 1e-6;
    double RS_L = 0.02;
    double RS_Vw = f.Vw[pos];

    //f.hState[pos] = RS_b * RS_V0 / RS_L * (exp((RS_f0 - state)/RS_b) - V/RS_V0);

    double Tau[3] = {0.,0.,0.};
    double Tau_n, Tau_s1, Tau_s2;// = f.Tn[j + k * nj];
    //Tau_n = -120e6;
    Tau_n = Ttilde_local[0];

    //real_t ifchange = 0; // false
    if(Ttilde_local[0] >= -1e-1){
      // fault can not open
      Tau_n = -1e-1;
      //ifchange = 1;
    }else{
      Tau_n = Ttilde_local[0];
    }

    Vtilde_local[0] = c*Ttilde_local[0];
    Vtilde_local[1] = c*Ttilde_local[1];
    Vtilde_local[2] = c*Ttilde_local[2];

//#ifdef TP
    real_t Pf;
    if(par.Friction_type == 3){
      Pf = f.TP_P[j+k*nj+0*nj*nk];
      //Pf = 0;
      // it is effective normal stress
      //Ttilde_local[0] += Pf;
      //T0_local[0] += Pf;
      Tau_n += Pf;
    }
//#endif

    double xnew, err, x, coef;
    int iter;

    coef = fabs(2.0*(dt1*Tau_n)/(rho*DH*jac)*RS_a);
#ifdef TractionImg
    coef = c*RS_a*fabs(Tau_n);
#endif

    // search absolute V 
    // or search V1 and V2
#define SearchAbsolute
//#define SearchEach

//   #ifdef SearchEach
//       // search in the eta direction
//       x = asinh( exp(state/RS_a) * Rate_local[1]/(2.0*RS_V0) );
//       NRsearch(&xnew, &iter, &err, x, state, RS_a, Vtilde_local[1], dt1, coef, 0);
//       //NRsearch(&xnew, &iter, &err, x, state, RS_a, Vhat_y, DT, coef, 0);
//       //if(j==nj/2 && k==nk/2) {
//       //  printf("Vhat=%.10f,%.10f,%.10f\n",Vhat_x,Vhat_y,Vhat_z);
//       //  printf("Vtil=%.10f,%.10f,%.10f\n", Vtilde_local[0],Vtilde_local[1],Vtilde_local[2]);
//       //  printf("vec_n=%g,%g,%g\n",vec_n[0],vec_n[1],vec_n[2]);
//       //  printf("vec_s1=%g,%g,%g\n",vec_s1[0],vec_s1[1],vec_s1[2]);
//       //  printf("vec_s2=%g,%g,%g\n",vec_s2[0],vec_s2[1],vec_s2[2]);
//       //}
//       
//       Ratey = 2.0*RS_V0*exp(-state/RS_a)*sinh(xnew);
//       Tau_s1 = RS_a * xnew * fabs(Tau_n);
//       // search in the zeta direction
//       // ===========================================================================
//       x = asinh( exp(state/RS_a) * Rate_local[2]/(2.0*RS_V0) );
//       NRsearch(&xnew, &iter, &err, x, state, RS_a, Vtilde_local[2], dt1, coef, 0);
//       Ratez = 2.0*RS_V0*exp(-state/RS_a)*sinh(xnew);
//       Tau_s2 = RS_a * xnew * fabs(Tau_n);
//   
//       V = sqrt(Ratey*Ratey+Ratez*Ratez);
//   #endif


#ifdef SearchAbsolute
    double Vtilde_local_norm = sqrt(
        Vtilde_local[1]*Vtilde_local[1]+Vtilde_local[2]*Vtilde_local[2]);
    double Ttilde_local_norm = sqrt(
        Ttilde_local[1]*Ttilde_local[1]+Ttilde_local[2]*Ttilde_local[2]);

    double Rate_norm = sqrt(
        Rate_local[1]*Rate_local[1]+Rate_local[2]*Rate_local[2]);

    x = asinh( exp((double)state/RS_a) * Rate_norm/(2.0*RS_V0) );
//#define Trapz
//  #ifdef Trapz
//      coef = 0.5*c*RS_a*fabs(Tau_n);
//      Vtilde_local_norm -= 0.5 * c * RS_a * x * fabs(Tau_n);
//  #endif
    NRsearch(&xnew, &iter, &err, x, state, RS_a, Vtilde_local_norm, dt1, coef, 0);
    V = 2.0*RS_V0*exp(-(double)state/RS_a)*sinh((double)xnew);
    double T = RS_a * xnew * fabs(Tau_n + 0);

    f.friction[j+k*nj] = RS_a * xnew;
    //if(j==nj/2 && k==(nk-1-int(7.5e3/DH))) {
    //  //printf("irk=%d,j=%d,k=%d,Vtil=%e,a=%e,v0=%e,x=%e,xnew=%e,T=%e,V=%e\n", irk,j,k, Vtilde_local_norm, RS_a, RS_V0, x, xnew, T, V);
    //  printf("irk=%d,j=%d,k=%d,Vtil=%e,psi=%e,a=%e,v0=%e,Rate_norm=%e,x=%e\n", irk,j,k, Vtilde_local_norm,state, RS_a, RS_V0, Rate_norm, x);
    //}

    //Vtilde_local_norm = max(Vtilde_local_norm, 1e-30);
    //Tau_s1 = Vtilde_local[1]/Vtilde_local_norm*T;
    //Tau_s2 = Vtilde_local[2]/Vtilde_local_norm*T;
    Ttilde_local_norm = max(Ttilde_local_norm, 1e-8);
    Tau_s1 = Ttilde_local[1]/Ttilde_local_norm*T;
    Tau_s2 = Ttilde_local[2]/Ttilde_local_norm*T;
#endif

    V = max(V, 1e-30);

//    // ageing law
//#if defined TPV101 || defined TPV102
//    RS_f0 = 0.6;
//    RS_V0 = 1e-6;
//    RS_L = 0.02;
//    f.hState[pos] = RS_b * RS_V0 / RS_L * (exp((RS_f0 - state)/RS_b) - V/RS_V0);
//#endif
//#if defined TPV103 || defined TPV104
//    RS_f0 = 0.6;
//    RS_V0 = 1e-6;
//    RS_L = 0.4;
//    double RS_fw = 0.2;
//
//    double RS_flv = RS_f0 - (RS_b-RS_a)*log(V/RS_V0);
//    double RS_fss = RS_fw + (RS_flv - RS_fw)/pow((1.+pow(V/RS_Vw, 8)),0.125);
//    double psiss = RS_a*(log(sinh(RS_fss/RS_a)) + log(2.*(RS_V0/V)));
//
//    f.hState[pos] = -V/RS_L*(state-psiss);
//    //f.State[pos] = (f.State[pos]-psiss)*exp(-V*DT/RS_L) + psiss;
//#endif
//
//    RS_f0 = par.f0;
//    RS_V0 = par.V0;
//    RS_L = par.L;
//    RS_L = f.L[j+k*nj];
//    if(par.Friction_type == 1){
//      f.hState[pos] = RS_b * RS_V0 / RS_L * (exp((RS_f0 - state)/RS_b) - V/RS_V0);
//    }else if (par.Friction_type == 2){
//      double RS_fw = par.fw;
//      double RS_flv = RS_f0 - (RS_b-RS_a)*log(V/RS_V0);
//      double RS_fss = RS_fw + (RS_flv - RS_fw)/pow((1.+pow(V/RS_Vw, 8)),0.125);
//      double psiss = RS_a*(log(sinh(RS_fss/RS_a)) + log(2.*(RS_V0/V)));
//      f.hState[pos] = -V/RS_L*(state-psiss);
//    }

    //if(j == nj/2 && k == nk/2){
    //  printf("Tau = %e %e %e, friction = %e, Tn = %e\n",
    //      Tau[0], Tau[1], Tau[2], RS_a * xnew, Tau_n);
    //}

    //pos = (j1 + k1 * ny) * 3;

    //double vec_s1[3], vec_s2[3], vec_n[3];
    //vec_s1[0] = f.vec_s1[pos + 0];
    //vec_s1[1] = f.vec_s1[pos + 1];
    //vec_s1[2] = f.vec_s1[pos + 2];
    //vec_s2[0] = f.vec_s2[pos + 0];
    //vec_s2[1] = f.vec_s2[pos + 1];
    //vec_s2[2] = f.vec_s2[pos + 2];

    //pos = j1 + k1 * ny + i0 * ny * nz;
    //vec_n[0] = XIX[pos];//M[pos + 0];
    //vec_n[1] = XIY[pos];//M[pos + 1];
    //vec_n[2] = XIZ[pos];//M[pos + 2];
    //vec_n0 = norm3(vec_n);

    //jacvec = JAC[pos] * vec_n0;

    //for (int ii = 0; ii < 3; ++ii){
    //  vec_n[ii] /= vec_n0;
    //}

    //pos1 = j1 + k1 * ny + nyz;
    pos1 = j1 + k1 * ny + 3*nyz;
    pos  = j + k * nj;
//#ifdef TP
    if(par.Friction_type == 3){
      Tau_n -= Pf; // not effective normal stress
    }
//#endif

    // transform back to x, y, z
    Tau[0] = Tau_n * vec_n[0] + Tau_s1 * vec_s1[0] + Tau_s2 * vec_s2[0];
    Tau[1] = Tau_n * vec_n[1] + Tau_s1 * vec_s1[1] + Tau_s2 * vec_s2[1];
    Tau[2] = Tau_n * vec_n[2] + Tau_s1 * vec_s1[2] + Tau_s2 * vec_s2[2];
    //Tau[0] = Tau_n
    Tau[0] = f.str_init_x[pos]; // force the Tn perturb to 0

    f.T11[pos1] = (Tau[0] - f.str_init_x[pos])*jacvec;
    //f.T11[pos1] = 0;
    f.T12[pos1] = (Tau[1] - f.str_init_y[pos])*jacvec;
    f.T13[pos1] = (Tau[2] - f.str_init_z[pos])*jacvec;

    real_t hT11, hT12, hT13;
    real_t viscosity = par.viscosity * DT;

    real_t DT2 = DT;
    if(irk == 0){
      DT2 = 0*DT;
    }else if(irk == 1){
      DT2 = 0.5*DT;
    }else if(irk == 2){
      DT2 = 0.5*DT;
    }else if(irk == 3){
      DT2 = 1.0*DT;
    }

    if(irk==0){
      hT11 = f.hT11[j1+k1*ny];
      hT12 = f.hT12[j1+k1*ny];
      hT13 = f.hT13[j1+k1*ny];
    }else{
      // update
      hT11 = (f.T11[pos1] - f.mT11[pos1])/DT2;
      hT12 = (f.T12[pos1] - f.mT12[pos1])/DT2;
      hT13 = (f.T13[pos1] - f.mT13[pos1])/DT2;
    }
    hT11 = (f.T11[pos1] - f.mT11[pos1])/DT;
    hT12 = (f.T12[pos1] - f.mT12[pos1])/DT;
    hT13 = (f.T13[pos1] - f.mT13[pos1])/DT;

#ifndef DxV_hT1
    if(irk==3){
    f.hT11[j1+k1*ny] = hT11;
    f.hT12[j1+k1*ny] = hT12;
    f.hT13[j1+k1*ny] = hT13;
    }
#endif
    //if(f.slip[j1+k1*ny] > par.Dc){
    //f.hT11[j1+k1*ny] = 0;//hT11;
    //f.hT12[j1+k1*ny] = 0;//hT12;
    //f.hT13[j1+k1*ny] = 0;//hT13;
    //}

    f.T11[pos1] += viscosity * hT11;
    f.T12[pos1] += viscosity * hT12;
    f.T13[pos1] += viscosity * hT13;

    pos = j + k * nj;

    //Tau[0] = f.T11[pos1];
    //Tau[1] = f.T12[pos1];
    //Tau[2] = f.T13[pos1];

    //f.tTs1[pos] = dot_product(Tau, vec_s1);
    //f.tTs2[pos] = dot_product(Tau, vec_s2);
    //f.tTn [pos] = dot_product(Tau, vec_n);
    f.tTs1[pos] = DOT3(Tau, vec_s1);
    f.tTs2[pos] = DOT3(Tau, vec_s2);
    f.tTn [pos] = DOT3(Tau, vec_n);
    //f.tTn [pos] = Tau_n;
    //f.tTn [pos] = -120e6;

    pos = j + k * nj;
    if(!f.init_t0_flag[pos]) {
      if (V > 1e-3) {
        f.init_t0[pos] = it * DT;
        f.init_t0_flag[pos] = 1;
        f.flag_rup[pos] = 1;
      }
    }

#ifdef SelectStencil
    pos = j + k*nj;
    if(f.flag_rup[pos] && f.first_rup[pos]){
      for (int l = -3; l <=3; l++){
        int jj1 = j+l;
        jj1 = max(0, jj1);
        jj1 = min(nj-1, jj1);
        int pos2 = (jj1+k*nj);
        f.rup_index_y[pos2] += 1;
        int kk1 = k+l;
        kk1 = max(0, kk1);
        kk1 = min(nk-1, kk1);
        pos2 = (j+kk1*nj);
        f.rup_index_z[pos2] += 1;
      }
      f.first_rup[pos] = 0;
    }
#endif
  } // end j k
  return;
}

void trial_rs(Wave W,Fault F,real_t *M,int it,int irk,int FlagX,int FlagY,int FlagZ)
{
  dim3 block(16, 8, 1);
  dim3 grid(
      (hostParams.nj+block.x-1)/block.x,
      (hostParams.nk+block.y-1)/block.y, 1);
  trial_rs_cu <<<grid, block>>> (W, F, M, it, irk, FlagX, FlagY, FlagZ);
}
