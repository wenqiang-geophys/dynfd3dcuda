
#include <hip/hip_runtime.h>
#ifdef useNetCDF
#include <stdio.h>
#include <stdlib.h>
//#include "netcdf.h"
#include "params.h"
#include "common.h"
#include "io.h"

void nc_def_wave_xy(int global_k, ncFile *nc)
{
  int err;
  int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;

  int dimz = (int) (global_k / nk);

  if (thisid[2] != dimz) return;

  char filename[1000];
  sprintf(filename, "%s/wave_xy_mpi%02d%02d%02d.nc", OUT,
      thisid[0], thisid[1], thisid[2]);
  err = nc_create(filename, NC_CLOBBER, &(nc->ncid)); handle_err(err);

  // define dimensions
  err = nc_def_dim(nc->ncid, "nt", NC_UNLIMITED, &(nc->dimid[0]));
  err = nc_def_dim(nc->ncid, "nx", ni,           &(nc->dimid[1]));
  err = nc_def_dim(nc->ncid, "ny", nj,           &(nc->dimid[2]));
  handle_err(err);

  const int dimid2[2] = {nc->dimid[1], nc->dimid[2]};

  // define variables
#ifdef DoublePrecision
  err = nc_def_var(nc->ncid, "x", NC_DOUBLE, 2, dimid2, &(nc->varid[20]));
  err = nc_def_var(nc->ncid, "y", NC_DOUBLE, 2, dimid2, &(nc->varid[21]));
  err = nc_def_var(nc->ncid, "z", NC_DOUBLE, 2, dimid2, &(nc->varid[22]));

  err = nc_def_var(nc->ncid, "Vx" , NC_DOUBLE, 3, nc->dimid, &nc->varid[0]);
  err = nc_def_var(nc->ncid, "Vy" , NC_DOUBLE, 3, nc->dimid, &nc->varid[1]);
  err = nc_def_var(nc->ncid, "Vz" , NC_DOUBLE, 3, nc->dimid, &nc->varid[2]);
#else
  err = nc_def_var(nc->ncid, "x", NC_FLOAT, 2, dimid2, &(nc->varid[20]));
  err = nc_def_var(nc->ncid, "y", NC_FLOAT, 2, dimid2, &(nc->varid[21]));
  err = nc_def_var(nc->ncid, "z", NC_FLOAT, 2, dimid2, &(nc->varid[22]));

  err = nc_def_var(nc->ncid, "Vx" , NC_FLOAT, 3, nc->dimid, &nc->varid[0]);
  err = nc_def_var(nc->ncid, "Vy" , NC_FLOAT, 3, nc->dimid, &nc->varid[1]);
  err = nc_def_var(nc->ncid, "Vz" , NC_FLOAT, 3, nc->dimid, &nc->varid[2]);
#endif

  // end define
  err = nc_enddef(nc->ncid);
  handle_err(err);

  return;
}

void nc_put_wave_xy_coord(real_t *C, int global_k, ncFile nc)
{
  int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;

  int dimz = (int) (global_k / nk);
  int k = global_k - dimz * nk;

  if (thisid[2] != dimz) return;

  int nx = ni + 6;
  int ny = nj + 6;
  int nz = nk + 6;

  real_t *x = (real_t *) malloc(sizeof(real_t)*ni*nj);
  real_t *y = (real_t *) malloc(sizeof(real_t)*ni*nj);
  real_t *z = (real_t *) malloc(sizeof(real_t)*ni*nj);

  for (int i = 0; i < ni; i++){
    for (int j = 0; j < nj; j++){
      int j1 = j + 3;
      int k1 = k + 3;
      int i1 = i + 3;
      int pos = j1 + k1 * ny + i1 * ny * nz;
      int nxyz = nx*ny*nz;
      x[j + i * nj] = C[pos + 0 * nxyz];
      y[j + i * nj] = C[pos + 1 * nxyz];
      z[j + i * nj] = C[pos + 2 * nxyz];
    }
  }

  int err;
#ifdef DoublePrecision
  err = nc_put_var_double(nc.ncid, nc.varid[20], x);
  err = nc_put_var_double(nc.ncid, nc.varid[21], y);
  err = nc_put_var_double(nc.ncid, nc.varid[22], z);
#else
  err = nc_put_var_float(nc.ncid, nc.varid[20], x);
  err = nc_put_var_float(nc.ncid, nc.varid[21], y);
  err = nc_put_var_float(nc.ncid, nc.varid[22], z);
#endif
  handle_err(err);

  free(x);free(y);free(z);
  return;
}

__global__ void get_wave_slice_z(real_t *W, int k, real_t *W1)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  int ni = par.ni;
  int nj = par.nj;

  int ny = par.ny;
  int nz = par.nz;

  if (i < ni && j < nj){
    int i1 = i+3;
    int j1 = j+3;
    int k1 = k+3;
    int pos = j1 + k1 * ny + i1 * ny * nz;
    W1[j + i * nj] = W[pos];
  }
  return;
}

void nc_put_wave_xy(real_t *W, int global_k, int it, ncFile nc)
{
  int err;

  int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;

  int nx = hostParams.nx;
  int ny = hostParams.ny;
  int nz = hostParams.nz;

  long nxyz = nx*ny*nz;

  int dimz = (int) (global_k / nk);
  int k = global_k - dimz * nk;

  if (thisid[2] != dimz) return;

  dim3 block(16, 16, 1);
  dim3 grid( (nj + block.x - 1 )/block.x, (ni + block.y - 1 )/block.y, 1);

  real_t *hostData, *deviceData;
  size_t size = sizeof(real_t)*ni*nj;

  cudaMalloc((real_t **) &deviceData, size);
  hostData = (real_t *) malloc(size);

  size_t start[3] = {it, 0, 0};
  size_t count[3] = {1, ni, nj};
  for (int ivar = 0; ivar < 3; ivar ++){
    get_wave_slice_z <<< grid, block >>> (W + ivar*nxyz, k, deviceData);
    cudaMemcpy(hostData, deviceData, size, cudaMemcpyDeviceToHost);
#ifdef DoublePrecision
    err = nc_put_vara_double(nc.ncid, nc.varid[ivar], start, count, hostData);
#else
    err = nc_put_vara_float(nc.ncid, nc.varid[ivar], start, count, hostData);
#endif
    handle_err(err);
  }

  nc_sync(nc.ncid);

  free(hostData);
  cudaFree(deviceData);
  return;
}

void nc_end_wave_xy(int global_k, ncFile nc)
{
  int nk = hostParams.nk;
  int dimz = (int) (global_k / nk);
  if (thisid[2] != dimz) return;

  int err;
  nc_sync(nc.ncid);
  err = nc_close(nc.ncid);
  handle_err(err);

  return;
}
#endif
