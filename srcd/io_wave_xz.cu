
#include <hip/hip_runtime.h>
#ifdef useNetCDF
#include <stdio.h>
#include <stdlib.h>
//#include "netcdf.h"
#include "params.h"
#include "common.h"
#include "io.h"

void nc_def_wave_xz(int global_j, ncFile *nc)
{
  int err;
  int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;

  int dimy = (int) (global_j / nj);

  if (thisid[1] != dimy) return;

  char filename[1000];
  sprintf(filename, "%s/wave_xz_mpi%02d%02d%02d.nc", OUT,
      thisid[0], thisid[1], thisid[2]);
  err = nc_create(filename, NC_CLOBBER, &(nc->ncid)); handle_err(err);

  // define dimensions
  err = nc_def_dim(nc->ncid, "nt", NC_UNLIMITED, &(nc->dimid[0]));
  err = nc_def_dim(nc->ncid, "nx", ni,           &(nc->dimid[1]));
  err = nc_def_dim(nc->ncid, "nz", nk,           &(nc->dimid[2]));
  handle_err(err);

  const int dimid2[2] = {nc->dimid[1], nc->dimid[2]};

  // define variables
#ifdef DoublePrecision
  err = nc_def_var(nc->ncid, "x", NC_DOUBLE, 2, dimid2, &(nc->varid[20]));
  err = nc_def_var(nc->ncid, "y", NC_DOUBLE, 2, dimid2, &(nc->varid[21]));
  err = nc_def_var(nc->ncid, "z", NC_DOUBLE, 2, dimid2, &(nc->varid[22]));

  err = nc_def_var(nc->ncid, "Vx" , NC_DOUBLE, 3, nc->dimid, &nc->varid[0]);
  err = nc_def_var(nc->ncid, "Vy" , NC_DOUBLE, 3, nc->dimid, &nc->varid[1]);
  err = nc_def_var(nc->ncid, "Vz" , NC_DOUBLE, 3, nc->dimid, &nc->varid[2]);
#else
  err = nc_def_var(nc->ncid, "x", NC_FLOAT, 2, dimid2, &(nc->varid[20]));
  err = nc_def_var(nc->ncid, "y", NC_FLOAT, 2, dimid2, &(nc->varid[21]));
  err = nc_def_var(nc->ncid, "z", NC_FLOAT, 2, dimid2, &(nc->varid[22]));

  err = nc_def_var(nc->ncid, "Vx" , NC_FLOAT, 3, nc->dimid, &nc->varid[0]);
  err = nc_def_var(nc->ncid, "Vy" , NC_FLOAT, 3, nc->dimid, &nc->varid[1]);
  err = nc_def_var(nc->ncid, "Vz" , NC_FLOAT, 3, nc->dimid, &nc->varid[2]);
#endif

  // end define
  err = nc_enddef(nc->ncid);
  handle_err(err);

  return;
}

void nc_put_wave_xz_coord(real_t *C, int global_j, ncFile nc)
{
  int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;

  int dimy = (int) (global_j / nj);
  int j = global_j - dimy * nj;

  if (thisid[1] != dimy) return;

  int nx = ni + 6;
  int ny = nj + 6;
  int nz = nk + 6;

  real_t *x = (real_t *) malloc(sizeof(real_t)*ni*nk);
  real_t *y = (real_t *) malloc(sizeof(real_t)*ni*nk);
  real_t *z = (real_t *) malloc(sizeof(real_t)*ni*nk);

  for (int i = 0; i < ni; i++){
    for (int k = 0; k < nk; k++){
      int j1 = j + 3;
      int k1 = k + 3;
      int i1 = i + 3;
      int pos = j1 + k1 * ny + i1 * ny * nz;
      int nxyz = nx*ny*nz;
      x[k + i * nk] = C[pos + 0 * nxyz];
      y[k + i * nk] = C[pos + 1 * nxyz];
      z[k + i * nk] = C[pos + 2 * nxyz];
    }
  }

  int err;
#ifdef DoublePrecision
  err = nc_put_var_double(nc.ncid, nc.varid[20], x);
  err = nc_put_var_double(nc.ncid, nc.varid[21], y);
  err = nc_put_var_double(nc.ncid, nc.varid[22], z);
#else
  err = nc_put_var_float(nc.ncid, nc.varid[20], x);
  err = nc_put_var_float(nc.ncid, nc.varid[21], y);
  err = nc_put_var_float(nc.ncid, nc.varid[22], z);
#endif
  handle_err(err);

  free(x);free(y);free(z);
  return;
}

__global__ void get_wave_slice_y(real_t *W, int j, real_t *W1)
{
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  int ni = par.ni;
  int nk = par.nk;

  int ny = par.ny;
  int nz = par.nz;

  if (i < ni && k < nk){
    int i1 = i+3;
    int j1 = j+3;
    int k1 = k+3;
    int pos = j1 + k1 * ny + i1 * ny * nz;
    W1[k + i * nk] = W[pos];
  }
  return;
}

void nc_put_wave_xz(real_t *W, int global_j, int it, ncFile nc)
{
  int err;

  int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;

  int nx = hostParams.nx;
  int ny = hostParams.ny;
  int nz = hostParams.nz;

  long nxyz = nx*ny*nz;

  int dimy = (int) (global_j / nj);
  int j = global_j - dimy * nj;

  if (thisid[1] != dimy) return;

  dim3 block(16, 16, 1);
  dim3 grid( (nk + block.x - 1 )/block.x, (ni + block.y - 1 )/block.y, 1);

  real_t *hostData, *deviceData;
  size_t size = sizeof(real_t)*ni*nk;

  cudaMalloc((real_t **) &deviceData, size);
  hostData = (real_t *) malloc(size);

  size_t start[3] = {it, 0, 0};
  size_t count[3] = {1, ni, nk};
  for (int ivar = 0; ivar < 3; ivar ++){
    get_wave_slice_y <<< grid, block >>> (W + ivar*nxyz, j, deviceData);
    cudaMemcpy(hostData, deviceData, size, cudaMemcpyDeviceToHost);
#ifdef DoublePrecision
    err = nc_put_vara_double(nc.ncid, nc.varid[ivar], start, count, hostData);
#else
    err = nc_put_vara_float(nc.ncid, nc.varid[ivar], start, count, hostData);
#endif
    handle_err(err);
  }

  nc_sync(nc.ncid);

  free(hostData);
  cudaFree(deviceData);
  return;
}

void nc_end_wave_xz(int global_j, ncFile nc)
{
  int nj = hostParams.nj;
  int dimy = (int) (global_j / nj);
  if (thisid[1] != dimy) return;

  int err;
  nc_sync(nc.ncid);
  err = nc_close(nc.ncid);
  handle_err(err);

  return;
}
#endif
